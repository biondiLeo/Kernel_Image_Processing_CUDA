﻿#include <iostream>
#include <chrono>
#include <string>
#include <omp.h>
#include "image_processor.h"
#include "filter_kernel.h"
#include "performance_test.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>

// Definizione delle stringhe di comando
#define CMD_GAUSSIAN     "gaussian"
#define CMD_SHARPEN      "sharpen"
#define CMD_EDGE         "edge"
#define CMD_LAPLACIAN    "laplacian"
#define CMD_DOG          "dog"

#define CMD_CUDA_GLOBAL  "global"
#define CMD_CUDA_CONST   "constant"
#define CMD_CUDA_SHARED  "shared"

#define OUTPUT_DIR       "output/"
#define IMG_EXT         ".png"

int main(int argc, char** argv) {
    // Verifica parametri di input
    if (argc < 3) {
        std::cerr << "Utilizzo: " << argv[0] << " tipo_filtro percorso_immagine [tipo_memoria_cuda]" << std::endl;
        std::cerr << "tipo_filtro: <gaussian | sharpen | edge | laplacian | dog>" << std::endl;
        std::cerr << "percorso_immagine: percorso del file immagine" << std::endl;
        std::cerr << "(opzionale) tipo_memoria_cuda: <global | constant | shared> (default: constant)" << std::endl;
        return 1;
    }

    std::string cmdFilter = std::string(argv[1]);
    std::string imagePath = std::string(argv[2]);
    std::string cudaMemType = (argc > 3) ? std::string(argv[3]) : CMD_CUDA_CONST;

    // Crea e configura il filtro
    FilterKernel filter;
    if (cmdFilter == CMD_GAUSSIAN) {
        filter.createGaussianFilter(7, 1.0f);
    }
    else if (cmdFilter == CMD_SHARPEN) {
        filter.createSharpeningFilter();
    }
    else if (cmdFilter == CMD_EDGE) {
        filter.createEdgeDetectionFilter();
    }
    else if (cmdFilter == CMD_LAPLACIAN) {
        filter.createLaplacianFilter();
    }
    else if (cmdFilter == CMD_DOG) {
        filter.createDoGFilter();
    }
    else {
        std::cerr << "Tipo di filtro non valido: " << cmdFilter << std::endl;
        return 1;
    }
    // Mostra il filtro selezionato
    std::cout << "\nFiltro selezionato: " << cmdFilter << std::endl;
    filter.displayKernel();

    // Carica l'immagine
    ImageProcessor inputImage;
    if (!inputImage.loadImageFromFile(imagePath.c_str())) {
        std::cerr << "Errore nel caricamento dell'immagine: " << imagePath << std::endl;
        return 1;
    }

    // Crea le immagini di output
    ImageProcessor outputCUDA;
    ImageProcessor outputCPU;
    std::vector<ImageProcessor> outputsOpenMP;

    // === Sezione Elaborazione Sequenziale ===
    std::cout << "\n=== Versione Sequenziale ===" << std::endl;
    auto t1 = std::chrono::high_resolution_clock::now();
    bool cpuResult = inputImage.applyFilterSequential(outputCPU, filter);
    auto t2 = std::chrono::high_resolution_clock::now();

    if (cpuResult) {
        auto cpuDuration = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count();
        std::cout << "Tempo di esecuzione CPU: " << cpuDuration << " microsec" << std::endl;
        std::string outputCpuPath = OUTPUT_DIR + std::string("cpu_") + cmdFilter + IMG_EXT;
        outputCPU.saveImageToFile(outputCpuPath.c_str());
    }

    // === Sezione Elaborazione CUDA ===
    std::cout << "\n=== Versione Parallela CUDA ===" << std::endl;
    // Determina il tipo di memoria CUDA da usare
    CudaMemoryType memType = CudaMemoryType::CONSTANT_MEM;
    if (cudaMemType == CMD_CUDA_GLOBAL) {
        memType = CudaMemoryType::GLOBAL_MEM;
    }
    else if (cudaMemType == CMD_CUDA_SHARED) {
        memType = CudaMemoryType::SHARED_MEM;
    }

    // Inizializza CUDA
    hipFree(0);
    auto t3 = std::chrono::high_resolution_clock::now();
    bool cudaResult = inputImage.applyFilterParallel(outputCUDA, filter, memType);
    auto t4 = std::chrono::high_resolution_clock::now();

    if (cudaResult) {
        auto cudaDuration = std::chrono::duration_cast<std::chrono::microseconds>(t4 - t3).count();
        std::cout << "Tempo di esecuzione CUDA: " << cudaDuration << " microsec" << std::endl;
        std::string outputCudaPath = OUTPUT_DIR + std::string("cuda_") + cmdFilter + IMG_EXT;
        outputCUDA.saveImageToFile(outputCudaPath.c_str());
    }

    // === Sezione Elaborazione OpenMP ===
    std::cout << "\n=== Versione Parallela OpenMP ===" << std::endl;
    int maxThreads = omp_get_max_threads();
    std::cout << "Numero massimo di thread disponibili: " << maxThreads << std::endl;

    for (int numThreads = 2; numThreads <= maxThreads; numThreads *= 2) {
        ImageProcessor outputOMP;
        auto t5 = std::chrono::high_resolution_clock::now();
        bool ompResult = inputImage.applyFilterOpenMP(outputOMP, filter, numThreads);
        auto t6 = std::chrono::high_resolution_clock::now();

        if (ompResult) {
            auto ompDuration = std::chrono::duration_cast<std::chrono::microseconds>(t6 - t5).count();
            std::cout << "Tempo di esecuzione OpenMP (" << numThreads << " threads): "
                << ompDuration << " microsec" << std::endl;

            std::string outputOmpPath = OUTPUT_DIR + std::string("omp") +
                std::to_string(numThreads) + "_" +
                cmdFilter + IMG_EXT;
            outputOMP.saveImageToFile(outputOmpPath.c_str());
            outputsOpenMP.push_back(outputOMP);
        }
    }

    // === Sezione Test di Performance ===
    std::cout << "\n=== Test di Performance ===" << std::endl;
    runPerformanceTests("", filter);  // I parametri non vengono più usati ma manteniamo la compatibilità

    return 0;
}